#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdint.h>
#include <algorithm>
#include <unordered_map>
typedef unsigned long long uint64;
typedef unsigned int uint32;
typedef unsigned char uint8;

using namespace std;

#define mix(a,b,c) \
{ \
  a -= b; a -= c; a ^= (c>>13); \
  b -= c; b -= a; b ^= (a<<8); \
  c -= a; c -= b; c ^= (b>>13); \
  a -= b; a -= c; a ^= (c>>12);  \
  b -= c; b -= a; b ^= (a<<16); \
  c -= a; c -= b; c ^= (b>>5); \
  a -= b; a -= c; a ^= (c>>3);  \
  b -= c; b -= a; b ^= (a<<10); \
  c -= a; c -= b; c ^= (b>>15); \
}

#define MIN(a,b)  ((a)>(b)?(b):(a))

#define NEW(a,b)  ((a)<(b)?(b):(a))


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
    }
#endif
    return result;
}

// BOB hash
__host__ __device__
uint32 hash0(uint32 key) {
        //register ub4 a,b,c,len;
    uint32 a,b,c;
    uint32 len = 4;
    char* str = (char*)&key;
    //  uint32_t initval = 0;
    /* Set up the internal state */
    //len = length;
    a = b = 0x9e3779b9;  /* the golden ratio; an arbitrary value */
    c = 8311;//prime32[1000];         /* the previous hash value */

    /*---------------------------------------- handle most of the key */
    while (len >= 12)
    {
        a += (str[0] +((uint32)str[1]<<8) +((uint32)str[2]<<16) +((uint32)str[3]<<24));
        b += (str[4] +((uint32)str[5]<<8) +((uint32)str[6]<<16) +((uint32)str[7]<<24));
        c += (str[8] +((uint32)str[9]<<8) +((uint32)str[10]<<16)+((uint32)str[11]<<24));
        mix(a,b,c);
        str += 12; len -= 12;
    }

    /*------------------------------------- handle the last 11 bytes */
    c += len;
    switch(len)              /* all the case statements fall through */
    {
        case 11: c+=((uint32)str[10]<<24);
        // fall through
        case 10: c+=((uint32)str[9]<<16);
        // fall through
        case 9 : c+=((uint32)str[8]<<8);
        /* the first byte of c is reserved for the length */
        // fall through
        case 8 : b+=((uint32)str[7]<<24);
        // fall through
        case 7 : b+=((uint32)str[6]<<16);
        // fall through
        case 6 : b+=((uint32)str[5]<<8);
        // fall through
        case 5 : b+=str[4];
        // fall through
        case 4 : a+=((uint32)str[3]<<24);
        // fall through
        case 3 : a+=((uint32)str[2]<<16);
        // fall through
        case 2 : a+=((uint32)str[1]<<8);
        // fall through
        case 1 : a+=str[0];
        /* case 0: nothing left to add */
    }
    mix(a,b,c);
    /*-------------------------------------------- report the result */
    return c;
}


const int LEN = 36000;
const int BLOCK_SIZE = 1;
const int FIRST_LAYER_BUCKET = 16384;
const int LAST_LAYER_BUCKET = 512;
const int L1 = 2;
const int L2 = 2;


 
__device__ uint32 kernel_bucket_key[LEN];
__device__ uint32 kernel_bucket_timestamp[LEN];
__device__ uint32 kernel_bucket_freq[LEN];
__device__ uint32 mutex[LAST_LAYER_BUCKET];     //2^9 protect the same last layer
__device__ uint32 stamp;
uint32 bucket_key[LEN];
uint32 bucket_timestamp[LEN];
uint32 bucket_freq[LEN];


// read in data
uint32 * read_keys(char *filename,uint32* len){
    FILE* fp=fopen(filename,"rb");
    fseek(fp,0L,SEEK_END);
    uint32 size = ftell(fp);
    rewind(fp);
    if(size % 13){
        printf("wrong file size: %d\n",size);
        fclose(fp);
        return NULL;
    }
    *len = size/13;
    printf("size:  %d\n",*len);
    uint32 * keys;
    checkCuda(hipHostMalloc((void **)&keys,sizeof(uint32 )* (*len)));
    for(int i=0;i< *len;++i){
        char tmp[13];
        fread(tmp,13,sizeof(char),fp);
        *(keys+i)=*(uint32*)tmp;
    }
    fclose(fp);
    return keys;
}

// init memory
__global__ void kernel_init() {
    for (int i = 0; i < LEN; ++i)
        kernel_bucket_timestamp[i] = 0;

    for (int i = 0; i < LEN; ++i)
        kernel_bucket_freq[i] = 0;

    for (int i = 0; i < LAST_LAYER_BUCKET; ++i)
        mutex[i] = 0;
}


//   insert into AHeap
__device__ inline void insert_bucket(uint32  key,uint32 freq,uint32 timestamp){
    uint32 hash_val= hash0(key);
    uint32 hash_pos= hash_val % FIRST_LAYER_BUCKET;
    uint32* mutex_pos = (mutex + ((hash_val % FIRST_LAYER_BUCKET)>>(L1 +L2 -1)));
    uint32 layer_start_pos = 0;
    uint32 layer_bucket_num = FIRST_LAYER_BUCKET;
    bool next=true;
    int cnt=40; 
    while(next&&cnt--){
        int v=atomicCAS(mutex_pos,0,1);
    if(v==0){
    // d-left part insert
    for(int i=0;i< L1;++i){
        if(kernel_bucket_freq[layer_start_pos + hash_pos]== 0){
            kernel_bucket_timestamp[layer_start_pos+hash_pos] = timestamp;
            kernel_bucket_key[layer_start_pos+hash_pos] = key;
            kernel_bucket_freq[layer_start_pos+hash_pos] = freq;
            atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
            return;
        }
        if(kernel_bucket_key[layer_start_pos+hash_pos]==key){
            uint32 kernel_in_timestamp = kernel_bucket_timestamp[layer_start_pos + hash_pos];
            kernel_bucket_timestamp[layer_start_pos+hash_pos]= NEW(kernel_in_timestamp,timestamp);
            kernel_bucket_freq[layer_start_pos+hash_pos]+=freq;
            atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
            return;
        }
        else{
            uint32 kernel_in_timestamp = kernel_bucket_timestamp[layer_start_pos + hash_pos]; 
            uint32 kernel_in_freq = kernel_bucket_freq[layer_start_pos+hash_pos];
            if(timestamp==NEW(kernel_in_timestamp,timestamp)&&kernel_in_freq==1){
                kernel_bucket_timestamp[layer_start_pos+hash_pos] = timestamp;
                kernel_bucket_key[layer_start_pos+hash_pos]=key;
                kernel_bucket_freq[layer_start_pos+hash_pos] = freq;
                atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
                return;
            }
            if(timestamp==NEW(kernel_in_timestamp,timestamp)&& kernel_in_timestamp + 10000 < timestamp){
                kernel_bucket_timestamp[layer_start_pos+hash_pos] = timestamp;
                kernel_bucket_key[layer_start_pos+hash_pos]=key;
                kernel_bucket_freq[layer_start_pos+hash_pos] = freq;
                atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
                return;
            } 
            layer_start_pos += layer_bucket_num;
            layer_bucket_num /= 2;
            hash_pos /= 2;
        }
    }
    //  winner-tree part insert 
    for(int i=0;i< L2-1;++i){
        if(kernel_bucket_freq[layer_start_pos + hash_pos]== 0){
            kernel_bucket_timestamp[layer_start_pos+hash_pos] = timestamp;
            kernel_bucket_key[layer_start_pos+hash_pos]=key;
            kernel_bucket_freq[layer_start_pos+hash_pos] = freq;
            atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
            return;
        }
        if(kernel_bucket_key[layer_start_pos+hash_pos]==key){
            uint32 kernel_in_timestamp = kernel_bucket_timestamp[layer_start_pos + hash_pos];
            kernel_bucket_timestamp[layer_start_pos+hash_pos]= NEW(kernel_in_timestamp,timestamp);
            kernel_bucket_freq[layer_start_pos+hash_pos]+=freq;
            atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
            return;
        }
        else{
           // uint32 kernel_insert_weight = freq;
            uint32 kernel_in_timestamp= kernel_bucket_timestamp[layer_start_pos+hash_pos];
            uint32 kernel_in_freq=kernel_bucket_freq[layer_start_pos+hash_pos];
            uint32 kernel_in_key;
            kernel_in_key=kernel_bucket_key[layer_start_pos+hash_pos];
          //  uint32 kernel_in_weight = kernel_in_timestamp+kernel_in_freq;
            if( freq > kernel_in_freq){
                kernel_bucket_key[layer_start_pos+hash_pos]=key;
                kernel_bucket_timestamp[layer_start_pos+hash_pos]=timestamp;
                kernel_bucket_freq[layer_start_pos+hash_pos]=freq;
                key=kernel_in_key;
                freq = kernel_in_freq;
                timestamp =kernel_in_timestamp;
            }
            layer_start_pos += layer_bucket_num;
            layer_bucket_num /= 2;
            hash_pos /= 2; 
        }
    }
        next = false;
        atomicExch(mutex+ ((hash_val% FIRST_LAYER_BUCKET)>>(L1+L2-1)),0);
        
        
        return;
    }
    }
    return;

}


// kernel function, allocate packet to insert
__global__ void kernel_insert(uint32 * kernel_keys,uint32 start_timestamps, uint32 len){
    const uint32 id=threadIdx.x+blockIdx.x *blockDim.x;
    if (id >= len ) return;
    uint32  keys_at_bucket=kernel_keys[id];
    uint32 freq = 1;
    uint32 timestamp =  start_timestamps +id;
    insert_bucket(keys_at_bucket,freq,timestamp);
}

double expr_result[10];
int expr_pos=0;


// batch processing and multi-streaming
void insert(uint32 *keys,uint32 len){
    uint32 * kernel_keys;
    checkCuda(hipMalloc((void **)&kernel_keys,sizeof(uint32)*len));
        
     
    hipEvent_t startEvent,stopEvent;
    checkCuda( hipEventCreate(&startEvent));
    checkCuda( hipEventCreate(&stopEvent) );
    
    int batch_num = (len + BATCH_SIZE -1) / BATCH_SIZE;
    hipStream_t stream[batch_num];
    for(int i=0;i<batch_num;++i)
        checkCuda(hipStreamCreate(&stream[i]));
    


    checkCuda( hipEventRecord(startEvent,0));

    for(uint32 i=0, left=len; i< len;i+=BATCH_SIZE,left-=BATCH_SIZE ){
        uint32 size= MIN(left,BATCH_SIZE);
        checkCuda(hipMemcpyAsync(kernel_keys+i,keys+i,sizeof(uint32)*size,hipMemcpyHostToDevice,stream[i/BATCH_SIZE]));
    }
    for(uint32 i=0, left=len; i< len;i+=BATCH_SIZE,left-=BATCH_SIZE ){
        uint32 size= MIN(left,BATCH_SIZE);
        kernel_insert<<<(size+BLOCK_SIZE-1)/BLOCK_SIZE,BLOCK_SIZE,0,stream[i/BATCH_SIZE]>>>(kernel_keys+i,i,size);    
    }
    for(uint32 i=0;i<batch_num;++i)
        checkCuda(hipStreamSynchronize(stream[i]));
    checkCuda(hipEventRecord(stopEvent,0));
    checkCuda(hipEventSynchronize(stopEvent));
    
    float ms;
    checkCuda(hipEventElapsedTime(&ms,startEvent,stopEvent));

    expr_result[expr_pos++] = len/ms/1000;
    hipFree(kernel_keys);
    for(uint32 i=0;i<batch_num;++i)
    checkCuda(hipStreamDestroy(stream[i]));
    

}

uint32* true_result(uint32 * keys,uint32 * len){
    unordered_map<uint32,uint32> s;
    for(int i=0;i<*len;++i) ++s[keys[i]];
    *len=s.size();
    uint32* res=(uint32*)malloc(sizeof(uint32)*(*len));
    int pos=0;
    for(auto p:s){
        keys[pos]=p.first;
        res[pos]=p.second;
        ++pos;
    }
    return res;
}

// Result query
uint32* query(uint32* keys,uint32 len){
    uint32* res=(uint32*)malloc(sizeof(uint32)*len);
    checkCuda(hipMemcpyFromSymbol(bucket_key,HIP_SYMBOL(kernel_bucket_key),LEN*sizeof(uint32)));
    checkCuda(hipMemcpyFromSymbol(bucket_freq,HIP_SYMBOL(kernel_bucket_freq),LEN*sizeof(uint32)));
    for(int i=0;i<len;++i){
        res[i]=0;
        uint32 hash_val=hash0(keys[i]);
        uint32 layer_start_pos = 0;
        uint32 layer_bucket_num = FIRST_LAYER_BUCKET;
        uint32 hash_pos=hash_val% FIRST_LAYER_BUCKET;
        for(int j=0;j<L1+L2;++j){
            if(bucket_key[layer_start_pos+hash_pos]==keys[i])
                res[i]+=bucket_freq[layer_start_pos+hash_pos];
            hash_pos /= 2;
            layer_start_pos +=layer_bucket_num;
            layer_bucket_num /= 2;
        }
    }
    return res;
}

void compare(uint32 * real,uint32* my,uint32 len){
    int accuracy = 0;
    double are =0;
    for(int i=0;i<len;++i){
        accuracy += (real[i]==my[i]);
        are += ((real[i]>my[i])?(real[i]-my[i]):(my[i]-real[i]))/(double)real[i];
    }
//    printf("%d %lf\n",len,accuracy/(double)len);  
//Notice about decreasing dataset package number or increasing buckets in AHeap
}

void experiment_filename(char * filename){
    uint32 len;
    uint32* keys=read_keys(filename,&len);
    kernel_init<<<1,1>>>();
	insert(keys,len);
    uint32 * true_res=true_result(keys,&len);
    uint32 * my_res=query(keys,len);
    compare(true_res,my_res,len);
    free(my_res);
    free(true_res);
    hipHostFree(keys);

}

int main(){
    char filename[100];
    printf("%d ",BATCH_SIZE);
/**/    for(int i=0;i<10;++i){
        sprintf(filename,"../../data/5s/%d.dat",i);
        experiment_filename(filename);
    }/**/
    sort(expr_result,expr_result+10);
    printf("%lf %lf %lf Mpps\n",
        (expr_result[0]+expr_result[1])/2,
        (expr_result[4]+expr_result[5])/2,
        (expr_result[8]+expr_result[9])/2
    );
    return 0;
    
}
